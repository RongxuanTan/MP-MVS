#include "hip/hip_runtime.h"
#include "PatchMatch.h"


__device__ int Point2Idx(const int2& p, int width) 
{
	return p.y * width + p.x;
}

__device__ float Vec3DotVec3(const float4 vec1, const float4 vec2)
{
    return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__  void sort_small(float *d, const int n)
{
    int j;
    for (int i = 1; i < n; i++) {
        float tmp = d[i];
        for (j = i; j >= 1 && tmp < d[j-1]; j--)
            d[j] = d[j-1];
        d[j] = tmp;
    }
}

__device__  void setBit(unsigned int &input, const unsigned int n)
{
    input |= (unsigned int)(1 << n);
}

__device__  int isSet(unsigned int input, const unsigned int n)
{
    return (input >> n) & 1;
}

__device__ void Mat33DotVec3(const float mat[9], const float4 vec, float4 *result)
{
  result->x = mat[0] * vec.x + mat[1] * vec.y + mat[2] * vec.z;
  result->y = mat[3] * vec.x + mat[4] * vec.y + mat[5] * vec.z;
  result->z = mat[6] * vec.x + mat[7] * vec.y + mat[8] * vec.z;
}

__device__ void TransformPDFToCDF(float* probs, const int num_probs)
{
    float prob_sum = 0.0f;
    for (int i = 0; i < num_probs; ++i) {
        prob_sum += probs[i];
    }
    const float inv_prob_sum = 1.0f / prob_sum;

    float cum_prob = 0.0f;
    for (int i = 0; i < num_probs; ++i) {
        cum_prob += probs[i] * inv_prob_sum;
        probs[i] = cum_prob;
    }
    probs[num_probs-1] = 1.f;
}

__device__ int FindMinCostIndex(const float *costs, const int n)
{
    float min_cost = costs[0];
    int min_cost_idx = 0;
    for (int idx = 1; idx < n; ++idx) {
        if (costs[idx] <= min_cost) {
            min_cost = costs[idx];
            min_cost_idx = idx;
        }
    }
    return min_cost_idx;
}

__device__ float ComputeDepthfromPlaneHypothesis(const Camera camera, const float4 PlaneHypothesis, const int2 p)
{
    return -PlaneHypothesis.w * camera.K[0] / ((p.x - camera.K[2]) * PlaneHypothesis.x + (camera.K[0] / camera.K[4]) * (p.y - camera.K[5]) * PlaneHypothesis.y + camera.K[0] * PlaneHypothesis.z);
}

__device__ float4 TransformNormal(const Camera camera, float4 plane_hypothesis)
{
    float4 transformed_normal;
    transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[3] * plane_hypothesis.y + camera.R[6] * plane_hypothesis.z;
    transformed_normal.y = camera.R[1] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y + camera.R[7] * plane_hypothesis.z;
    transformed_normal.z = camera.R[2] * plane_hypothesis.x + camera.R[5] * plane_hypothesis.y + camera.R[8] * plane_hypothesis.z;
    transformed_normal.w = plane_hypothesis.w;
    return transformed_normal;
}
/*
在图形中，纹理是描述表面视觉外观的一组样本。样本是一个点。也就是说，它没有尺寸（与具有物理尺寸的像素相反）。
当使用样本来确定像素的颜色时，每个样本都位于其对应像素的中心。使用整数坐标寻址像素时，给定像素的精确中心将成为其整个数字坐标加上0.5（每个维度）的偏移量。
换句话说，将0.5添加到纹理坐标可确保从这些坐标读取时，返回该像素的样本的确切值。
但是，只有当纹理的filterMode设置为hipFilterModeLinear时，从纹理中读取的值才会在像素内发生变化。在该模式下，从不在像素的中心的坐标读取返回在给定像素的样本和相邻像素的样本之间插值的值。
所以，将0.5加到整数坐标有效地否定了hipFilterModeLinear模式。但是，由于在纹理坐标中添加0.5会占用内核中的周期，因此最好通过将filterMode设置为hipFilterModePoint来关闭内插。
然后，从像素内的任何坐标读取返回该像素的确切纹理采样值，因此可以使用整数直接读取纹理采样。
当使用hipFilterModePoint时，如果在计算纹理坐标时涉及浮点数学，必须注意确保浮点不准确不会导致纹理坐标超出预定目标像素的范围。
此外，正如评论提到的那样，代码中可能存在问题。将0.5f添加到纹理坐标意味着正在使用hipFilterModeLinear模式，但该模式返回一个浮点数，而不是一个int数。
*/
__device__ void TexProcessPixel(hipTextureObject_t *images,uchar *texMap,const int2& p,int width,int height, const PatchMatchParams params)
{
	if (p.x >= width || p.y >= height)
		return;
    float sum_ref=0.0f;
    float sum_ref2=0.0f;
    const int cows_n=(2*params.nSizeHalfWindow)/params.nSizeStep+1;
    const float inv_nm=1/(float)(cows_n*cows_n);
    for (int i = -params.nSizeHalfWindow; i <= params.nSizeHalfWindow; i += params.nSizeStep) {
        for (int j = -params.nSizeHalfWindow; j <= params.nSizeHalfWindow; j += params.nSizeStep) {
            const int2 pt=make_int2(p.x+i,p.y+j);
            const float refPix=tex2D<float>(images[0],pt.x+ 0.5f,pt.y+ 0.5f);
            sum_ref+=refPix;
            sum_ref2+=refPix*refPix;

        }
    }
    const float var_ref=std::abs(sum_ref2-sum_ref*sum_ref*inv_nm)*inv_nm;
    const int ind=Point2Idx(p,width);
    uchar cost=((uchar)var_ref)<50?0:((uchar)var_ref);
    texMap[ind]=cost<255?cost:255;
    // if(var_ref>100.0f)
    //     texMap[ind]=(uchar)(200);
}


__global__ void TextureConfMap(hipTextureObject_t *images,uchar *texMap,int width,int height, const PatchMatchParams params){
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    TexProcessPixel(images,texMap,p,width,height,params);
}
//no problem
__device__ void GetPointI2C(const Camera camera, const int2 p, const float depth, float *X)
{
    X[0] = depth * (p.x - camera.K[2]) / camera.K[0];
    X[1] = depth * (p.y - camera.K[5]) / camera.K[4];
    X[2] = depth;
}
//no problem
__device__ float GetPlane2Origin(const Camera camera, const int2 p, const float depth, const float4 normal)
{
    float X[3];
    GetPointI2C(camera, p, depth, X);
    return -(normal.x * X[0] + normal.y * X[1] + normal.z * X[2]);
}

//no problem 
__device__ float4 GetViewDirection(const Camera camera, const int2 p){
    float4 view_direction;
    view_direction.x = (p.x - camera.K[2]) / camera.K[0];
    view_direction.y = (p.y - camera.K[5]) / camera.K[4];
    view_direction.z = 1;
    view_direction.w = 0;
    return view_direction;
}

__device__ void NormalizeVec3 (float4 *vec)
{
    const float normSquared = vec->x * vec->x + vec->y * vec->y + vec->z * vec->z;
    const float inverse_sqrt = rsqrtf (normSquared);//平方根倒数
    vec->x *= inverse_sqrt;
    vec->y *= inverse_sqrt;
    vec->z *= inverse_sqrt;
}
//no problem 
__device__ float4 GenerateRandomNormal(const Camera camera, const int2 p, hiprandState *randState){
    float q1, q2, s;
	do {
		q1 = 2.f * hiprand_uniform(randState) - 1.f;
		q2 = 2.f * hiprand_uniform(randState) - 1.f;
		s = q1 * q1 + q2 * q2;
	} while (s >= 1.f);
	const float sq = sqrt(1.f - s);
    float4 normal;
    normal.x = 2.0f * q1 * sq;
    normal.y = 2.0f * q2 * sq;
    normal.z = 1.0f - 2.0f * s;
    normal.w = 0;
    const float4 view_direction = GetViewDirection(camera, p);
    float dot_product = normal.x * view_direction.x + normal.y * view_direction.y + normal.z * view_direction.z;
    if (dot_product > 0.0f) {
        normal.x = -normal.x;
        normal.y = -normal.y;
        normal.z = -normal.z;
    }
    NormalizeVec3(&normal);
    return normal;
}
//no problem 
__device__ float4 GenerateRandomPlaneHypothesis(const Camera camera, const int2 p, hiprandState *randState, const float depth_min, const float depth_max){
    float4 PlaneHypothesis = GenerateRandomNormal(camera, p, randState);//随机生成法向量
    float depth = hiprand_uniform(randState) * (depth_max - depth_min) + depth_min;//随机生成深度
    PlaneHypothesis.w = GetPlane2Origin(camera, p, depth, PlaneHypothesis);//计算平面到相机原点距离
    return PlaneHypothesis;
}

__device__ void ComputeHomography(const Camera ref_camera, const Camera src_camera, const float4 PlaneHypotheses, float *H)
{
    float R_relative[9];
    float C_relative[3];
    float t_relative[3];
    R_relative[0] = src_camera.R[0] * ref_camera.R[0] + src_camera.R[1] * ref_camera.R[1] + src_camera.R[2] *ref_camera.R[2];
    R_relative[1] = src_camera.R[0] * ref_camera.R[3] + src_camera.R[1] * ref_camera.R[4] + src_camera.R[2] *ref_camera.R[5];
    R_relative[2] = src_camera.R[0] * ref_camera.R[6] + src_camera.R[1] * ref_camera.R[7] + src_camera.R[2] *ref_camera.R[8];
    R_relative[3] = src_camera.R[3] * ref_camera.R[0] + src_camera.R[4] * ref_camera.R[1] + src_camera.R[5] *ref_camera.R[2];
    R_relative[4] = src_camera.R[3] * ref_camera.R[3] + src_camera.R[4] * ref_camera.R[4] + src_camera.R[5] *ref_camera.R[5];
    R_relative[5] = src_camera.R[3] * ref_camera.R[6] + src_camera.R[4] * ref_camera.R[7] + src_camera.R[5] *ref_camera.R[8];
    R_relative[6] = src_camera.R[6] * ref_camera.R[0] + src_camera.R[7] * ref_camera.R[1] + src_camera.R[8] *ref_camera.R[2];
    R_relative[7] = src_camera.R[6] * ref_camera.R[3] + src_camera.R[7] * ref_camera.R[4] + src_camera.R[8] *ref_camera.R[5];
    R_relative[8] = src_camera.R[6] * ref_camera.R[6] + src_camera.R[7] * ref_camera.R[7] + src_camera.R[8] *ref_camera.R[8];
    C_relative[0] = (ref_camera.C[0] - src_camera.C[0]);
    C_relative[1] = (ref_camera.C[1] - src_camera.C[1]);
    C_relative[2] = (ref_camera.C[2] - src_camera.C[2]);
    t_relative[0] = src_camera.R[0] * C_relative[0] + src_camera.R[1] * C_relative[1] + src_camera.R[2] * C_relative[2];
    t_relative[1] = src_camera.R[3] * C_relative[0] + src_camera.R[4] * C_relative[1] + src_camera.R[5] * C_relative[2];
    t_relative[2] = src_camera.R[6] * C_relative[0] + src_camera.R[7] * C_relative[1] + src_camera.R[8] * C_relative[2];

    H[0] = R_relative[0] - t_relative[0] * PlaneHypotheses.x / PlaneHypotheses.w;
    H[1] = R_relative[1] - t_relative[0] * PlaneHypotheses.y / PlaneHypotheses.w;
    H[2] = R_relative[2] - t_relative[0] * PlaneHypotheses.z / PlaneHypotheses.w;
    H[3] = R_relative[3] - t_relative[1] * PlaneHypotheses.x / PlaneHypotheses.w;
    H[4] = R_relative[4] - t_relative[1] * PlaneHypotheses.y / PlaneHypotheses.w;
    H[5] = R_relative[5] - t_relative[1] * PlaneHypotheses.z / PlaneHypotheses.w;
    H[6] = R_relative[6] - t_relative[2] * PlaneHypotheses.x / PlaneHypotheses.w;
    H[7] = R_relative[7] - t_relative[2] * PlaneHypotheses.y / PlaneHypotheses.w;
    H[8] = R_relative[8] - t_relative[2] * PlaneHypotheses.z / PlaneHypotheses.w;

    float tmp[9];
    tmp[0] = H[0] / ref_camera.K[0];
    tmp[1] = H[1] / ref_camera.K[4];
    tmp[2] = -H[0] * ref_camera.K[2] / ref_camera.K[0] - H[1] * ref_camera.K[5] / ref_camera.K[4] + H[2];
    tmp[3] = H[3] / ref_camera.K[0];
    tmp[4] = H[4] / ref_camera.K[4];
    tmp[5] = -H[3] * ref_camera.K[2] / ref_camera.K[0] - H[4] * ref_camera.K[5] / ref_camera.K[4] + H[5];
    tmp[6] = H[6] / ref_camera.K[0];
    tmp[7] = H[7] / ref_camera.K[4];
    tmp[8] = -H[6] * ref_camera.K[2] / ref_camera.K[0] - H[7] * ref_camera.K[5] / ref_camera.K[4] + H[8];

    H[0] = src_camera.K[0] * tmp[0] + src_camera.K[2] * tmp[6];
    H[1] = src_camera.K[0] * tmp[1] + src_camera.K[2] * tmp[7];
    H[2] = src_camera.K[0] * tmp[2] + src_camera.K[2] * tmp[8];
    H[3] = src_camera.K[4] * tmp[3] + src_camera.K[5] * tmp[6];
    H[4] = src_camera.K[4] * tmp[4] + src_camera.K[5] * tmp[7];
    H[5] = src_camera.K[4] * tmp[5] + src_camera.K[5] * tmp[8];
    H[6] = src_camera.K[8] * tmp[6];
    H[7] = src_camera.K[8] * tmp[7];
    H[8] = src_camera.K[8] * tmp[8];
}

__device__ float2 ComputeCorrespondingPoint(const float *H, const int2 p)
{
    float3 pt;
    pt.x = H[0] * p.x + H[1] * p.y + H[2];
    pt.y = H[3] * p.x + H[4] * p.y + H[5];
    pt.z = H[6] * p.x + H[7] * p.y + H[8];
    return make_float2(pt.x / pt.z, pt.y / pt.z);
}

__device__ float2 ComputeCorrespondingPointRow(const float *H, const float2 p)
{
    float3 pt;
    pt.x = p.x + H[0];
    pt.y = p.y + H[3];
    pt.z = 1 + H[6];
    return make_float2(pt.x / pt.z, pt.y / pt.z);
}

__device__ float2 ComputeCorrespondingPointCol(const float *H, const float2 p)
{
    float3 pt;
    pt.x = p.x + H[1];
    pt.y = p.y + H[4];
    pt.z = 1 + H[7];
    return make_float2(pt.x / pt.z, pt.y / pt.z);
}

__device__ float4 TransformNormal2RefCam(const Camera camera, float4 plane_hypothesis)
{
    float4 transformed_normal;
    transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[1] * plane_hypothesis.y + camera.R[2] * plane_hypothesis.z;
    transformed_normal.y = camera.R[3] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y + camera.R[5] * plane_hypothesis.z;
    transformed_normal.z = camera.R[6] * plane_hypothesis.x + camera.R[7] * plane_hypothesis.y + camera.R[8] * plane_hypothesis.z;
    transformed_normal.w = plane_hypothesis.w;
    return transformed_normal;
}

__device__ float ComputeBilateralWeight(const float x_dist, const float y_dist, const float pix, const float center_pix, const float sigma_spatial, const float sigma_color)
{
    const float spatial_dist = sqrt(x_dist * x_dist + y_dist * y_dist);
    const float color_dist = fabs(pix - center_pix);
    return exp(-spatial_dist / (2.0f * sigma_spatial* sigma_spatial) - color_dist / (2.0f * sigma_color * sigma_color));
}

__device__ float ComputeBilateralNCC(const hipTextureObject_t ref_image, const Camera ref_camera, const hipTextureObject_t src_image, const Camera src_camera, const int2 p, const float4 PlaneHypotheses, const PatchMatchParams params)
{
    // const float cost_max = 2.0f;
    // float H[9];
    // ComputeHomography(ref_camera, src_camera, PlaneHypotheses, H);
    // float2 pt = ComputeCorrespondingPoint(H, p);
    // if (pt.x >= src_camera.width || pt.x < 0.0f || pt.y >= src_camera.height || pt.y < 0.0f) {
    //     return cost_max;
    // }
    // int2 temp = make_int2(p.x-params.nSizeHalfWindow,p.y-params.nSizeHalfWindow);
    // float2 x = ComputeCorrespondingPoint(H, temp);
    // float2 basex(x);
    // for(int i=0;i<9;++i){
    //     H[i]=H[i]*(float)params.nSizeStep;
    // }

    const float cost_max = 2.0f;
    int radius = params.nSizeHalfWindow;

    float H[9];
    ComputeHomography(ref_camera, src_camera, PlaneHypotheses, H);
    float2 pt = ComputeCorrespondingPoint(H, p);
    if (pt.x >= src_camera.width || pt.x < 0.0f || pt.y >= src_camera.height || pt.y < 0.0f) {
        return cost_max;
    }

    float cost = 0.0f;
    {
        float sum_ref = 0.0f;
        float sum_ref_ref = 0.0f;
        float sum_src = 0.0f;
        float sum_src_src = 0.0f;
        float sum_ref_src = 0.0f;
        float bilateral_weight_sum = 0.0f;
        const float ref_center_pix = tex2D<float>(ref_image, p.x + 0.5f, p.y + 0.5f);

        for (int i = -radius; i < radius + 1; i += params.nSizeStep) {
            float sum_ref_row = 0.0f;
            float sum_src_row = 0.0f;
            float sum_ref_ref_row = 0.0f;
            float sum_src_src_row = 0.0f;
            float sum_ref_src_row = 0.0f;
            float bilateral_weight_sum_row = 0.0f;

            for (int j = -radius; j < radius + 1; j += params.nSizeStep) {
                const int2 ref_pt = make_int2(p.x + i, p.y + j);
                const float ref_pix = tex2D<float>(ref_image, ref_pt.x + 0.5f, ref_pt.y + 0.5f);
                float2 src_pt = ComputeCorrespondingPoint(H, ref_pt);
                const float src_pix = tex2D<float>(src_image, src_pt.x + 0.5f, src_pt.y + 0.5f);

                float weight = ComputeBilateralWeight(i, j, ref_pix, ref_center_pix, params.sigma_spatial, params.sigma_color);

                sum_ref_row += weight * ref_pix;
                sum_ref_ref_row += weight * ref_pix * ref_pix;
                sum_src_row += weight * src_pix;
                sum_src_src_row += weight * src_pix * src_pix;
                sum_ref_src_row += weight * ref_pix * src_pix;
                bilateral_weight_sum_row += weight;
            }

            sum_ref += sum_ref_row;
            sum_ref_ref += sum_ref_ref_row;
            sum_src += sum_src_row;
            sum_src_src += sum_src_src_row;
            sum_ref_src += sum_ref_src_row;
            bilateral_weight_sum += bilateral_weight_sum_row;
        }
        const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
        sum_ref *= inv_bilateral_weight_sum;
        sum_ref_ref *= inv_bilateral_weight_sum;
        sum_src *= inv_bilateral_weight_sum;
        sum_src_src *= inv_bilateral_weight_sum;
        sum_ref_src *= inv_bilateral_weight_sum;

        const float var_ref = sum_ref_ref - sum_ref * sum_ref;
        const float var_src = sum_src_src - sum_src * sum_src;

        const float kMinVar = 1e-5f;
        if (var_ref < kMinVar || var_src < kMinVar) {
            return cost = cost_max;
        } else {
            const float covar_src_ref = sum_ref_src - sum_ref * sum_src;
            const float var_ref_src = sqrt(var_ref * var_src);
            return cost = max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
        }
    }

    // {
    //     float sum_ref = 0.0f;
    //     float sum_ref_ref = 0.0f;
    //     float sum_src = 0.0f;
    //     float sum_src_src = 0.0f;
    //     float sum_ref_src = 0.0f;
    //     float bilateral_weight_sum = 0.0f;
    //     const float ref_center_pix = tex2D<float>(ref_image, p.x + 0.5f, p.y + 0.5f);

    //     for (int i = -params.nSizeHalfWindow; i <= params.nSizeHalfWindow; i += params.nSizeStep) {
    //         for (int j = -params.nSizeHalfWindow; j <= params.nSizeHalfWindow; j += params.nSizeStep) {
    //             const int2 ref_pt = make_int2(p.x + j, p.y + i);
    //             const float ref_pix = tex2D<float>(ref_image, ref_pt.x + 0.5f, ref_pt.y + 0.5f);
    //             const float2 src_pt = x;
    //             const float src_pix = tex2D<float>(src_image, src_pt.x + 0.5f, src_pt.y + 0.5f);
    //             float weight = ComputeBilateralWeight(j, i, ref_pix, ref_center_pix, params.sigma_spatial, params.sigma_color);
    //             const float weightRefPix = weight * ref_pix;
	// 		    const float weightSrcPix = weight * src_pix;
    //             sum_ref += weightRefPix;
    //             sum_ref_ref += weightRefPix * ref_pix;
    //             sum_src += weightSrcPix;
    //             sum_src_src += weightSrcPix * src_pix;
    //             sum_ref_src += weightRefPix* src_pix;
    //             bilateral_weight_sum += weight;
    //             x=ComputeCorrespondingPointRow(H, x);
    //         }
    //         basex=ComputeCorrespondingPointCol(H, basex);
    //         x=basex;
    //     }

    //     const float var_ref = sum_ref_ref * bilateral_weight_sum - sum_ref * sum_ref;
    //     const float var_src = sum_src_src * bilateral_weight_sum - sum_src * sum_src;

    //     const float kMinVar = 1e-5f;
    //     if (var_ref < kMinVar || var_src < kMinVar) {
    //         return cost_max;
    //     } else {
    //         const float covar_src_ref = sum_ref_src * bilateral_weight_sum - sum_ref * sum_src;
    //         const float var_ref_src = sqrt(var_ref * var_src);
    //         return max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
    //     }
    // }
}

__device__ float ComputeMultiViewInitialCostandSelectedViews(const hipTextureObject_t *images, const Camera *cameras, const int2 p, const float4 PlaneHypotheses, unsigned int *selected_views, const PatchMatchParams params)
{
    float cost_max = 2.0f;
    float cost_vector[32] = {2.0f};
    float cost_vector_copy[32] = {2.0f};
    int cost_count = 0;
    int num_valid_views = 0;

    for (int i = 1; i < params.num_images; ++i) {
        float c = ComputeBilateralNCC(images[0], cameras[0], images[i], cameras[i], p, PlaneHypotheses, params);
        cost_vector[i - 1] = c;
        cost_vector_copy[i - 1] = c;
        cost_count++;
        if (c < cost_max) {
            num_valid_views++;
        }
    }

    sort_small(cost_vector, cost_count);
    *selected_views = 0;

    int top_k = min(num_valid_views, params.top_k);
    if (top_k > 0) {
        float cost = 0.0f;
        for (int i = 0; i < top_k; ++i) {
            cost += cost_vector[i];
        }
        float cost_threshold = cost_vector[top_k - 1];
        for (int i = 0; i < params.num_images - 1; ++i) {
            if (cost_vector_copy[i] <= cost_threshold) {
                setBit(*selected_views, i);
            }
        }
        return cost / top_k;
    } else {
        return cost_max;
    }
}
//no problem 
__global__ void InitializeScore(const hipTextureObject_t* images, Camera* cameras, float4* PlaneHypotheses, float* costs, hiprandState* randStates, unsigned int *selected_views, const PatchMatchParams params){
    const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    int width = cameras[0].width;
    int height = cameras[0].height;

    if (p.x >= width || p.y >= height) {
        return;
    }

    const int idx = Point2Idx(p, width);
	hiprand_init(clock64(), p.y, p.x, &randStates[idx]);

    //随即生成平面假设
    if(!params.geom_consistency){
        PlaneHypotheses[idx]=GenerateRandomPlaneHypothesis(cameras[0], p, &randStates[idx], params.depth_min, params.depth_max);
        costs[idx] = ComputeMultiViewInitialCostandSelectedViews(images, cameras, p, PlaneHypotheses[idx], &selected_views[idx], params);
    }else{
        float4 plane_hypothesis=PlaneHypotheses[idx];
        plane_hypothesis = TransformNormal2RefCam(cameras[0], plane_hypothesis);
        float depth = plane_hypothesis.w;
        plane_hypothesis.w = GetPlane2Origin(cameras[0], p, depth, plane_hypothesis);
        PlaneHypotheses[idx] = plane_hypothesis;
        costs[idx] = ComputeMultiViewInitialCostandSelectedViews(images, cameras, p, PlaneHypotheses[idx], &selected_views[idx], params);
    }
    
}

__device__ void ComputeMultiViewCostVector(const hipTextureObject_t *images, const Camera *cameras, const int2 p, const float4 PlaneHypothesis, float *cost_vector, const PatchMatchParams params)
{
    for (int i = 1; i < params.num_images; ++i) {
        cost_vector[i - 1] = ComputeBilateralNCC(images[0], cameras[0], images[i], cameras[i], p, PlaneHypothesis, params);
    }
}

__device__ float4 GeneratePerturbedNormal(const Camera camera, const int2 p, const float4 normal, hiprandState *rand_state, const float perturbation)
{
    float4 view_direction = GetViewDirection(camera, p);

    const float a1 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
    const float a2 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
    const float a3 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;

    const float sin_a1 = sin(a1);
    const float sin_a2 = sin(a2);
    const float sin_a3 = sin(a3);
    const float cos_a1 = cos(a1);
    const float cos_a2 = cos(a2);
    const float cos_a3 = cos(a3);

    float R[9];
    R[0] = cos_a2 * cos_a3;
    R[1] = cos_a3 * sin_a1 * sin_a2 - cos_a1 * sin_a3;
    R[2] = sin_a1 * sin_a3 + cos_a1 * cos_a3 * sin_a2;
    R[3] = cos_a2 * sin_a3;
    R[4] = cos_a1 * cos_a3 + sin_a1 * sin_a2 * sin_a3;
    R[5] = cos_a1 * sin_a2 * sin_a3 - cos_a3 * sin_a1;
    R[6] = -sin_a2;
    R[7] = cos_a2 * sin_a1;
    R[8] = cos_a1 * cos_a2;

    float4 normal_perturbed;
    Mat33DotVec3(R, normal, &normal_perturbed);

    if (Vec3DotVec3(normal_perturbed, view_direction) >= 0.0f) {
        return normal;
    }

    NormalizeVec3(&normal_perturbed);
    return normal_perturbed;
}

__device__ float3 BackProjectPoint2W(const float x, const float y, const float depth, const Camera camera)
{
    // float3 pointX;
    // float3 tmpX;
    // // Reprojection
    // pointX.x = depth * (x - camera.K[2]) / camera.K[0];
    // pointX.y = depth * (y - camera.K[5]) / camera.K[4];
    // pointX.z = depth;

    // // Transformation
    // pointX.x = camera.R[0] * pointX.x + camera.R[3] * pointX.y + camera.R[6] * pointX.z + camera.C[0];
    // pointX.y = camera.R[1] * pointX.x + camera.R[4] * pointX.y + camera.R[7] * pointX.z + camera.C[1];
    // pointX.z = camera.R[2] * pointX.x + camera.R[5] * pointX.y + camera.R[8] * pointX.z + camera.C[2];

    // return pointX;
    float3 pointX;
    float3 tmpX;
    // Reprojection
    pointX.x = depth * (x - camera.K[2]) / camera.K[0];
    pointX.y = depth * (y - camera.K[5]) / camera.K[4];
    pointX.z = depth;

    // Rotation
    tmpX.x = camera.R[0] * pointX.x + camera.R[3] * pointX.y + camera.R[6] * pointX.z;
    tmpX.y = camera.R[1] * pointX.x + camera.R[4] * pointX.y + camera.R[7] * pointX.z;
    tmpX.z = camera.R[2] * pointX.x + camera.R[5] * pointX.y + camera.R[8] * pointX.z;

    // Transformation
    float3 C;
    C.x = -(camera.R[0] * camera.t[0] + camera.R[3] * camera.t[1] + camera.R[6] * camera.t[2]);
    C.y = -(camera.R[1] * camera.t[0] + camera.R[4] * camera.t[1] + camera.R[7] * camera.t[2]);
    C.z = -(camera.R[2] * camera.t[0] + camera.R[5] * camera.t[1] + camera.R[8] * camera.t[2]);
    pointX.x = tmpX.x + C.x;
    pointX.y = tmpX.y + C.y;
    pointX.z = tmpX.z + C.z;

    return pointX;

}

__device__ void ProjectPoint(const float3 PointX, const Camera camera, float2 &point, float &depth)
{
    // float3 tmp;
    // tmp.x = camera.R[0] * PointX.x + camera.R[1] * PointX.y + camera.R[2] * PointX.z + camera.t[0];
    // tmp.y = camera.R[3] * PointX.x + camera.R[4] * PointX.y + camera.R[5] * PointX.z + camera.t[1];
    // tmp.z = camera.R[6] * PointX.x + camera.R[7] * PointX.y + camera.R[8] * PointX.z + camera.t[2];

    // const float depth = camera.K[6] * tmp.x + camera.K[7] * tmp.y + camera.K[8] * tmp.z;
    // point.x = (camera.K[0] * tmp.x + camera.K[1] * tmp.y + camera.K[2] * tmp.z) / depth;
    // point.y = (camera.K[3] * tmp.x + camera.K[4] * tmp.y + camera.K[5] * tmp.z) / depth;
    float3 tmp;
    tmp.x = camera.R[0] * PointX.x + camera.R[1] * PointX.y + camera.R[2] * PointX.z + camera.t[0];
    tmp.y = camera.R[3] * PointX.x + camera.R[4] * PointX.y + camera.R[5] * PointX.z + camera.t[1];
    tmp.z = camera.R[6] * PointX.x + camera.R[7] * PointX.y + camera.R[8] * PointX.z + camera.t[2];

    depth = camera.K[6] * tmp.x + camera.K[7] * tmp.y + camera.K[8] * tmp.z;
    point.x = (camera.K[0] * tmp.x + camera.K[1] * tmp.y + camera.K[2] * tmp.z) / depth;
    point.y = (camera.K[3] * tmp.x + camera.K[4] * tmp.y + camera.K[5] * tmp.z) / depth;    
}

__device__ float3 Get3DPointonWorld_cu(const float x, const float y, const float depth, const Camera camera)
{
    float3 pointX;
    float3 tmpX;
    // Reprojection
    pointX.x = depth * (x - camera.K[2]) / camera.K[0];
    pointX.y = depth * (y - camera.K[5]) / camera.K[4];
    pointX.z = depth;

    // Rotation
    tmpX.x = camera.R[0] * pointX.x + camera.R[3] * pointX.y + camera.R[6] * pointX.z;
    tmpX.y = camera.R[1] * pointX.x + camera.R[4] * pointX.y + camera.R[7] * pointX.z;
    tmpX.z = camera.R[2] * pointX.x + camera.R[5] * pointX.y + camera.R[8] * pointX.z;

    // Transformation
    float3 C;
    C.x = -(camera.R[0] * camera.t[0] + camera.R[3] * camera.t[1] + camera.R[6] * camera.t[2]);
    C.y = -(camera.R[1] * camera.t[0] + camera.R[4] * camera.t[1] + camera.R[7] * camera.t[2]);
    C.z = -(camera.R[2] * camera.t[0] + camera.R[5] * camera.t[1] + camera.R[8] * camera.t[2]);
    pointX.x = tmpX.x + C.x;
    pointX.y = tmpX.y + C.y;
    pointX.z = tmpX.z + C.z;

    return pointX;
}

__device__ void ProjectonCamera_cu(const float3 PointX, const Camera camera, float2 &point, float &depth)
{
    float3 tmp;
    tmp.x = camera.R[0] * PointX.x + camera.R[1] * PointX.y + camera.R[2] * PointX.z + camera.t[0];
    tmp.y = camera.R[3] * PointX.x + camera.R[4] * PointX.y + camera.R[5] * PointX.z + camera.t[1];
    tmp.z = camera.R[6] * PointX.x + camera.R[7] * PointX.y + camera.R[8] * PointX.z + camera.t[2];

    depth = camera.K[6] * tmp.x + camera.K[7] * tmp.y + camera.K[8] * tmp.z;
    point.x = (camera.K[0] * tmp.x + camera.K[1] * tmp.y + camera.K[2] * tmp.z) / depth;
    point.y = (camera.K[3] * tmp.x + camera.K[4] * tmp.y + camera.K[5] * tmp.z) / depth;
}

__device__ float ComputeGeomConsistencyCost(const hipTextureObject_t depth_image, const Camera ref_camera, const Camera src_camera, const float4 PlaneHypothesis, const int2 p)
{
    const float max_cost = 3.0f;

    float depth = ComputeDepthfromPlaneHypothesis(ref_camera, PlaneHypothesis, p);
    float3 forward_point = Get3DPointonWorld_cu(p.x, p.y, depth, ref_camera);

    float2 src_pt;
    float src_d;
    ProjectonCamera_cu(forward_point, src_camera, src_pt, src_d);
    const float src_depth = tex2D<float>(depth_image,  (int)src_pt.x + 0.5f, (int)src_pt.y + 0.5f);

    if (src_depth == 0.0f) {
        return max_cost;
    }

    float3 src_3D_pt = Get3DPointonWorld_cu(src_pt.x, src_pt.y, src_depth, src_camera);

    float2 backward_point;
    float ref_d;
    ProjectonCamera_cu(src_3D_pt, ref_camera, backward_point, ref_d);

    const float diff_col = p.x - backward_point.x;
    const float diff_row = p.y - backward_point.y;
    return min(max_cost, sqrt(diff_col * diff_col + diff_row * diff_row));
}

__device__ void PlaneHypothesisRefinement(const hipTextureObject_t *images, const hipTextureObject_t *depth_images, const Camera *cameras, float4 *PlaneHypothesis, float *depth, float *cost, hiprandState *rand_state, const float *view_weights, const float weight_norm, float *restricted_cost, const int2 p, const PatchMatchParams params)
{
    float perturbation = 0.02f;
    float depth_rand;
    float4 PlaneHypothesisRand;

    depth_rand = hiprand_uniform(rand_state) * (params.depth_max - params.depth_min) + params.depth_min;
    PlaneHypothesisRand = GenerateRandomNormal(cameras[0], p, rand_state);

    float depth_perturbed = *depth;
    const float depth_min_perturbed = (1 - perturbation) * depth_perturbed;
    const float depth_max_perturbed = (1 + perturbation) * depth_perturbed;
    do {//在平面假设附近增加随机扰动
        depth_perturbed = hiprand_uniform(rand_state) * (depth_max_perturbed - depth_min_perturbed) + depth_min_perturbed;
    } while (depth_perturbed < params.depth_min && depth_perturbed > params.depth_max);
    float4 PlaneHypothesisPerturbed = GeneratePerturbedNormal(cameras[0], p, *PlaneHypothesis, rand_state, perturbation * M_PI);

    const int num_planes = 5;
    float depths[num_planes] = {depth_rand, *depth, depth_rand, *depth, depth_perturbed};
    float4 normals[num_planes] = {*PlaneHypothesis, PlaneHypothesisRand, PlaneHypothesisRand, PlaneHypothesisPerturbed, *PlaneHypothesis};

    for (int i = 0; i < num_planes; ++i) {
        float cost_vector[32] = {2.0f};
        float4 temp_plane_hypothesis = normals[i];
        temp_plane_hypothesis.w = GetPlane2Origin(cameras[0], p, depths[i], temp_plane_hypothesis);
        ComputeMultiViewCostVector(images, cameras, p, temp_plane_hypothesis, cost_vector, params);

        float temp_cost = 0.0f;
        for (int j = 0; j < params.num_images - 1; ++j) {
            if (view_weights[j] > 0) {
                if (params.geom_consistency) {
                    temp_cost += view_weights[j] * (cost_vector[j] + 0.2f * ComputeGeomConsistencyCost(depth_images[j+1], cameras[0], cameras[j+1], temp_plane_hypothesis, p));
                }
                else {
                    temp_cost += view_weights[j] * cost_vector[j];
                }
            }
        }
        temp_cost /= weight_norm;

        float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], temp_plane_hypothesis, p);

        if (depth_before >= params.depth_min && depth_before <= params.depth_max && temp_cost < *cost) {
            //*depth = depth_before;
            *PlaneHypothesis = temp_plane_hypothesis;
            *cost = temp_cost;
        }

    }
}

__device__ void CheckerboardPropagation(const hipTextureObject_t *images, const hipTextureObject_t *depths, const Camera *cameras, float4 *PlaneHypotheses, float *costs, float *pre_costs, hiprandState *randStates, unsigned int *selected_views, const int2 p, const PatchMatchParams params, const int iter)
{
    int width = cameras[0].width;
    int height = cameras[0].height;
    if (p.x >= width || p.y >= height) {
        return;
    }

    const int idx = Point2Idx(p, width);
    hiprandState* randState = &randStates[idx];
	float4& plane = PlaneHypotheses[idx];

	// adaptive sampling: 0 up-near, 1 down-near, 2 left-near, 3 right-near, 4 up-far, 5 down-far, 6 left-far, 7 right-far
	static constexpr int2 dirs[8][11] = {
		{{ 0,-1},{-1,-2},{ 1,-2},{-2,-3},{ 2,-3},{-3,-4},{ 3,-4}},
		{{ 0, 1},{-1, 2},{ 1, 2},{-2, 3},{ 2, 3},{-3, 4},{ 3, 4}},
		{{-1, 0},{-2,-1},{-2, 1},{-3,-2},{-3, 2},{-4,-3},{-4, 3}},
		{{ 1, 0},{ 2,-1},{ 2, 1},{ 3,-2},{ 3, 2},{ 4,-3},{ 4, 3}},
		{{0,-3},{0,-5},{0,-7},{0,-9},{0,-11},{0,-13},{0,-15},{0,-17},{0,-19},{0,-21},{0,-23}},
		{{0, 3},{0, 5},{0, 7},{0, 9},{0, 11},{0, 13},{0, 15},{0, 17},{0, 19},{0, 21},{0, 23}},
		{{-3,0},{-5,0},{-7,0},{-9,0},{-11,0},{-13,0},{-15,0},{-17,0},{-19,0},{-21,0},{-23,0}},
		{{ 3,0},{ 5,0},{ 7,0},{ 9,0},{ 11,0},{ 13,0},{ 15,0},{ 17,0},{ 19,0},{ 21,0},{ 23,0}}
	};
	static constexpr int numDirs[8] = {7, 7, 7, 7, 11, 11, 11, 11};
	const int neighborPositions[4] = {
		idx - width,
		idx + width,
		idx - 1,
		idx + 1,
	};
    int positions[8];
    float cost_array[8][32] = {2.0f};
    bool flag[8] = {false};

	for (int posId=0; posId<8; ++posId) {
		const int2* samples = dirs[posId];
		int2 bestNx; 
        float bestConf(FLT_MAX);
		for (int dirId=0; dirId<numDirs[posId]; ++dirId) {
			const int2& offset = samples[dirId];
			const int2 np=make_int2(p.x+offset.x, p.y+offset.y);
			if (!(np.x>=0 && np.y>=0 && np.x<width && np.y<height))
				continue;
			const int nidx = Point2Idx(np, width);
			const float nconf = costs[nidx];
			if (bestConf > nconf) {
				bestNx = np;
				bestConf = nconf;
			}
		}
		if (bestConf < FLT_MAX) {
			flag[posId]=true;
			positions[posId] = Point2Idx(bestNx, width);
			ComputeMultiViewCostVector(images, cameras, p, PlaneHypotheses[positions[posId]], cost_array[posId], params);
		}
	}

    // Multi-hypothesis Joint View Selection,Consider 4-neighborhood smoothness 
    float view_weights[32] = {0.0f};
    float view_selection_priors[32] = {0.0f};
    for (int i = 0; i < 4; ++i) {
        if (flag[i]) {
            for (int j = 0; j < params.num_images - 1; ++j) {
                view_selection_priors[j] += (isSet(selected_views[neighborPositions[i]], j) ? 0.9f : 0.1f);
            }
        }
    }

    float sampling_probs[32] = {0.0f};
    float cost_threshold = 0.8 * expf((iter) * (iter) / (-90.0f));
    for (int i = 0; i < params.num_images - 1; i++) {
        float count = 0;
        int count_false = 0;
        float tmpw = 0;
        for (int j = 0; j < 8; j++) {
            if (cost_array[j][i] < cost_threshold) {
                tmpw += expf(cost_array[j][i] * cost_array[j][i] / (-0.18f));
                count++;
            }
            if (cost_array[j][i] > 1.2f) {
                count_false++;
            }
        }
        if (count > 2 && count_false < 3) {
            sampling_probs[i] = view_selection_priors[i]*tmpw / count;
        }
        else if (count_false < 3) {
            sampling_probs[i] = view_selection_priors[i]*expf(cost_threshold * cost_threshold / (-0.32f));
        }else{
            sampling_probs[i] = 0;
        }
    }
    //PDF:probability density function  CDF:cumulative distribution function
    TransformPDFToCDF(sampling_probs, params.num_images - 1);
    for (int sample = 0; sample < 15; ++sample) {
        const float rand_prob = hiprand_uniform(&randStates[idx]) - FLT_EPSILON;

        for (int image_id = 0; image_id < params.num_images - 1; ++image_id) {
            const float prob = sampling_probs[image_id];
            if (prob > rand_prob) {
                view_weights[image_id] += 1.0f;
                break;
            }
        }
    }

    unsigned int temp_selected_views = 0;
    int num_selected_view = 0;
    float weight_norm = 0;
    for (int i = 0; i < params.num_images - 1; ++i) {
        if (view_weights[i] > 0) {
            setBit(temp_selected_views, i);
            weight_norm += view_weights[i];
            num_selected_view++;
        }
    }

    float final_costs[8] = {0.0f};
    for (int i = 0; i < 8; ++i) {
        for (int j = 0; j < params.num_images - 1; ++j) {
            if (view_weights[j] > 0) {
                if (params.geom_consistency) {
                    if (flag[i]) {
                        final_costs[i] += view_weights[j] * (cost_array[i][j] + 0.2f * ComputeGeomConsistencyCost(depths[j+1], cameras[0], cameras[j+1], PlaneHypotheses[positions[i]], p));
                    }
                    else {
                        final_costs[i] += view_weights[j] * (cost_array[i][j] + 0.1f * 3.0f);
                    }
                }
                else {
                    final_costs[i] += view_weights[j] * cost_array[i][j];
                }
            }
        }
        final_costs[i] /= weight_norm;
    }

    const int min_cost_idx = FindMinCostIndex(final_costs, 8);

    float cost_vector_now[32] = {2.0f};
    ComputeMultiViewCostVector(images, cameras, p, PlaneHypotheses[idx], cost_vector_now, params);
    float cost_now = 0.0f;
    for (int i = 0; i < params.num_images - 1; ++i) {
        if (params.geom_consistency) {
            cost_now += view_weights[i] * (cost_vector_now[i] + 0.2f * ComputeGeomConsistencyCost(depths[i+1], cameras[0], cameras[i+1], PlaneHypotheses[idx], p));
        }
        else {
            cost_now += view_weights[i] * cost_vector_now[i];
        }
    }
    cost_now /= weight_norm;
    costs[idx] = cost_now;
    float depth_now = ComputeDepthfromPlaneHypothesis(cameras[0], PlaneHypotheses[idx], p);
    float restricted_cost = 0.0f;

    float4 plane_hypotheses_now=PlaneHypotheses[idx];
    if (flag[min_cost_idx]) {
        float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], PlaneHypotheses[positions[min_cost_idx]], p);

        if (depth_before >= params.depth_min && depth_before <= params.depth_max && final_costs[min_cost_idx] < cost_now) {
            depth_now = depth_before;
           plane_hypotheses_now = PlaneHypotheses[positions[min_cost_idx]];
           cost_now = final_costs[min_cost_idx];
           selected_views[idx] = temp_selected_views;
        }
    }

    PlaneHypothesisRefinement(images, depths, cameras, &plane_hypotheses_now, &depth_now, &cost_now, &randStates[idx], view_weights, weight_norm, &restricted_cost, p, params);
    costs[idx] = cost_now;
    PlaneHypotheses[idx] = plane_hypotheses_now;

}

__global__ void BlackPixelUpdate(const hipTextureObject_t* images, const hipTextureObject_t *depths, Camera *cameras, float4 *PlaneHypotheses, float *costs, float *pre_costs, hiprandState *randStates, unsigned int *selected_views, const PatchMatchParams params, const int iter){
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2;
    } else {
        p.y = p.y * 2 + 1;
    }
    CheckerboardPropagation(images, depths, cameras, PlaneHypotheses, costs, pre_costs, randStates, selected_views, p, params, iter);

}

__global__ void RedPixelUpdate(const hipTextureObject_t* images, const hipTextureObject_t *depths, Camera *cameras, float4 *PlaneHypotheses, float *costs, float *pre_costs, hiprandState *randStates, unsigned int *selected_views, const PatchMatchParams params, const int iter){
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2 + 1;
    } else {
        p.y = p.y * 2;
    }
    CheckerboardPropagation(images, depths, cameras, PlaneHypotheses, costs, pre_costs, randStates, selected_views, p, params, iter);
}

__global__ void GetDepthandNormal(Camera *cameras, float4 *plane_hypotheses, const PatchMatchParams params)
{
    const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int width = cameras[0].width;
    const int height = cameras[0].height;

    if (p.x >= width || p.y >= height) {
        return;
    }

    const int center = p.y * width + p.x;
    plane_hypotheses[center].w = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);
    plane_hypotheses[center] = TransformNormal(cameras[0], plane_hypotheses[center]);
}

__device__ void CheckerboardFilter(const Camera *cameras, float4 *plane_hypotheses, float *costs, const int2 p)
{
    int width = cameras[0].width;
    int height = cameras[0].height;
    if (p.x >= width || p.y >= height) {
        return;
    }

    const int center = p.y * width + p.x;

    float filter[21];
    int index = 0;

    filter[index++] = plane_hypotheses[center].w;

    // Left
    const int left = center - 1;
    const int leftleft = center - 3;

    // Up
    const int up = center - width;
    const int upup = center - 3 * width;

    // Down
    const int down = center + width;
    const int downdown = center + 3 * width;

    // Right
    const int right = center + 1;
    const int rightright = center + 3;

    if (costs[center] < 0.001f) {
        return;
    }

    if (p.y>0) {
        filter[index++] = plane_hypotheses[up].w;
    }
    if (p.y>2) {
        filter[index++] = plane_hypotheses[upup].w;
    }
    if (p.y>4) {
        filter[index++] = plane_hypotheses[upup-width*2].w;
    }
    if (p.y<height-1) {
        filter[index++] = plane_hypotheses[down].w;
    }
    if (p.y<height-3) {
        filter[index++] = plane_hypotheses[downdown].w;
    }
    if (p.y<height-5) {
        filter[index++] = plane_hypotheses[downdown+width*2].w;
    }
    if (p.x>0) {
        filter[index++] = plane_hypotheses[left].w;
    }
    if (p.x>2) {
        filter[index++] = plane_hypotheses[leftleft].w;
    }
    if (p.x>4) {
        filter[index++] = plane_hypotheses[leftleft-2].w;
    }
    if (p.x<width-1) {
        filter[index++] = plane_hypotheses[right].w;
    }
    if (p.x<width-3) {
        filter[index++] = plane_hypotheses[rightright].w;
    }
    if (p.x<width-5) {
        filter[index++] = plane_hypotheses[rightright+2].w;
    }
    if (p.y>0 &&
        p.x<width-2) {
        filter[index++] = plane_hypotheses[up+2].w;
    }
    if (p.y< height-1 &&
        p.x<width-2) {
        filter[index++] = plane_hypotheses[down+2].w;
    }
    if (p.y>0 &&
        p.x>1)
    {
        filter[index++] = plane_hypotheses[up-2].w;
    }
    if (p.y<height-1 &&
        p.x>1) {
        filter[index++] = plane_hypotheses[down-2].w;
    }
    if (p.x>0 &&
        p.y>2)
    {
        filter[index++] = plane_hypotheses[left  - width*2].w;
    }
    if (p.x<width-1 &&
        p.y>2)
    {
        filter[index++] = plane_hypotheses[right - width*2].w;
    }
    if (p.x>0 &&
        p.y<height-2) {
        filter[index++] = plane_hypotheses[left  + width*2].w;
    }
    if (p.x<width-1 &&
        p.y<height-2) {
        filter[index++] = plane_hypotheses[right + width*2].w;
    }

    sort_small(filter,index);
    int median_index = index / 2;
    if (index % 2 == 0) {
        plane_hypotheses[center].w = (filter[median_index-1] + filter[median_index]) / 2;
    } else {
        plane_hypotheses[center].w = filter[median_index];
    }
}

__global__ void BlackPixelFilter(const Camera *cameras, float4 *plane_hypotheses, float *costs)
{
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2;
    } else {
        p.y = p.y * 2 + 1;
    }

    CheckerboardFilter(cameras, plane_hypotheses, costs, p);
}

__global__ void RedPixelFilter(const Camera *cameras, float4 *plane_hypotheses, float *costs)
{
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2 + 1;
    } else {
        p.y = p.y * 2;
    }

    CheckerboardFilter(cameras, plane_hypotheses, costs, p);
}


void PatchMatchCUDA::Run(){
    const int width=cameras[0].width;
    const int height=cameras[0].height;

    int BLOCK_W = 32;
    int BLOCK_H = (BLOCK_W / 2);
    const dim3 blockSize(BLOCK_W,BLOCK_H,1);
    const dim3 gridSizeInit((width+BLOCK_W-1)/BLOCK_W,(height+BLOCK_H-1)/BLOCK_H,1);
	const dim3 gridSizeCheckerboard((width + BLOCK_W - 1) / BLOCK_W, ((height / 2) + BLOCK_H - 1) / BLOCK_H, 1);

    int max_iterations = params.max_iterations;

    //计算纹理系数
    //TextureConfMap<<<gridSizeInit,blockSize>>>(cudaTextureImages,cudaTexCofMap,width,height,params);
    //hipDeviceSynchronize();
    //深度图初始化
    InitializeScore<<<gridSizeInit,blockSize>>>(cudaTextureImages,cudaCameras,cudaPlaneHypotheses,cudaCosts,cudaRandStates,cudaSelectedViews,params);
    hipDeviceSynchronize();
    if(params.geom_consistency){
    for (int i = 0; i < max_iterations; ++i) {
        BlackPixelUpdate<<<gridSizeCheckerboard, blockSize>>>(cudaTextureImages, cudaTextureDepths, cudaCameras, cudaPlaneHypotheses, cudaCosts, cudaPreCosts, cudaRandStates, cudaSelectedViews, params, i);
        checkCudaCall(hipDeviceSynchronize());
        RedPixelUpdate<<<gridSizeCheckerboard, blockSize>>>(cudaTextureImages, cudaTextureDepths, cudaCameras, cudaPlaneHypotheses, cudaCosts, cudaPreCosts,cudaRandStates, cudaSelectedViews, params, i);
        checkCudaCall(hipDeviceSynchronize());
        printf("iteration: %d/%d\n", i+1,max_iterations);
    }
    }
    else{
            for(int scale=2;scale>=0;--scale)
            {
                params.nSizeStep=(int)pow(2.0,scale+1);
                params.nSizeHalfWindow=params.nSizeStep*5/2;
                printf("Scale: %d\n", scale);
                for (int i = 0; i < max_iterations; ++i) {
                    BlackPixelUpdate<<<gridSizeCheckerboard, blockSize>>>(cudaTextureImages, cudaTextureDepths, cudaCameras, cudaPlaneHypotheses, cudaCosts, cudaPreCosts, cudaRandStates, cudaSelectedViews, params, i);
                    checkCudaCall(hipDeviceSynchronize());
                    RedPixelUpdate<<<gridSizeCheckerboard, blockSize>>>(cudaTextureImages, cudaTextureDepths, cudaCameras, cudaPlaneHypotheses, cudaCosts, cudaPreCosts,cudaRandStates, cudaSelectedViews, params, i);
                    checkCudaCall(hipDeviceSynchronize());
                    printf("iteration: %d/%d\n", i+1,max_iterations);
                }
            }
    }


    GetDepthandNormal<<<gridSizeInit,blockSize>>>(cudaCameras, cudaPlaneHypotheses, params);
    checkCudaCall(hipDeviceSynchronize());

    BlackPixelFilter<<<gridSizeCheckerboard, blockSize>>>(cudaCameras, cudaPlaneHypotheses, cudaCosts);
    checkCudaCall(hipDeviceSynchronize());
    RedPixelFilter<<<gridSizeCheckerboard, blockSize>>>(cudaCameras, cudaPlaneHypotheses, cudaCosts);
    checkCudaCall(hipDeviceSynchronize());


    checkCudaCall(hipMemcpy(hostPlaneHypotheses, cudaPlaneHypotheses, sizeof(float4) * width * height, hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(hostCosts, cudaCosts, sizeof(float) * width * height, hipMemcpyDeviceToHost));
    // std::cout<<"init secuss"<<std::endl;
    // checkCudaCall(hipMemcpy(hostPlaneHypotheses, cudaPlaneHypotheses, sizeof(float4)*cameras[0].width*cameras[0].height, hipMemcpyDeviceToHost));
    // std::cout<<"normal read"<<std::endl;
    // NormalVisualize(hostPlaneHypotheses,width,height);
    // cv::Mat img_gray(height, width, CV_8UC1);
    // hipMemcpy(img_gray.data, cudaTexCofMap, sizeof(uchar)*width*height, hipMemcpyDeviceToHost);
    // std::cout<<"hello"<<std::endl;
    // cv::imwrite("/home/xuan/MP-MVS/dense/result.jpg",img_gray);

}
